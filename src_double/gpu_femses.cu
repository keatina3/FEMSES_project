#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <cassert>
#include <cstdio>
#include <vector>
#include "mesh.h"
#include "utils.h"
#include "gpu_utils.h"
#include "gpu_fem.h"
#include "gpu_femses.h"

//////////// Calculates weighting for assembling single element solution ///////////
// One weight is evaluated for each node
// Added back to global memory
__device__ void calc_weights(double *w, int *cells, double *temp1, int idx, int idy){
    double *Le;
    int v;
    int offset = 28*threadIdx.x;

    Le = &temp1[offset];
    v = cells[(idx*3) + idy];

    atomicAddDouble(&w[v], Le[(idy*3) + idy]);
}
////////


/////////// Copies element matrices/element vector from global-shared memory //////////
__device__ void elems_glob_cpy(double *Le, double *be, double *temp1, int idx, int idy){
    double *Le_shrd, *be_shrd;
    int offset = 28*threadIdx.x;

    Le_shrd = &temp1[offset];
    be_shrd = &temp1[offset + 9];

    be[(idx*3) + idy] = be_shrd[idy];

    for(int i=0; i<3; i++){
        Le[(idx*9) + (idy*3) + i] = Le_shrd[(idy*3) + i];
    }
}
////////


/////////// Copies element matrices/element vector from shared-global memory //////////
__device__ void elems_shared_cpy(double *Le, double *be, double *temp1, int idx, int idy){
    double *Le_shrd, *be_shrd;
    int offset = 15*threadIdx.x;

    Le_shrd = &temp1[offset];
    be_shrd = &temp1[offset + 9];

    be_shrd[idy] = be[(idx*3) + idy];
    for(int i=0; i<3; i++){
        Le_shrd[(idy*3) + i] = Le[(idx*9) + (idy*3) + i];
    }
}
////////


////////////// Performs Jacobi iteration to get updated approximation of u ////////////
__device__ void jacobi_iter(
                double *ue,
                double *up_glob,
                int *cells,
                double *temp1,
                int idx,
                int idy)
{
    double *Le_shrd, *be_shrd;
    double ue_new, *ue_old;
    int v;
    int offset = 15*threadIdx.x;

    Le_shrd = &temp1[offset];
    be_shrd = &temp1[offset + 9];
    ue_old  = &temp1[offset + 12];

    v = cells[(idx*3) + idy];

    ue_new = be_shrd[idy];
    ue_old[idy] = up_glob[v];

    __syncthreads();
    
    ue_new -= Le_shrd[(idy*3) + ((idy+1)%3) ] * ue_old[ (idy+1) % 3];
    ue_new -= Le_shrd[(idy*3) + ((idy+2)%3) ] * ue_old[ (idy+2) % 3];

    ue_new /= Le_shrd[(idy*3) + idy];

    // atomicExch(&ue[(idx*3) + idy], ue_new); // transferring element solution of u to global mem
    ue[(idx*3) + idy] = ue_new;
}
//////


/////////////////// Kernel to assemble element solutions ///////////////////////////
// Element solutions are calculated in shared memory
// Element solutions are then transferred to an array in global memory
__global__ void assemble_elems_gpu(
                double *Le, 
                double *be, 
                double *w,
                double *u_glob,
                double *vertices, 
                int *cells, 
                int *is_bound, 
                double *bdry_vals,
                int order,
                int num_cells)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;      // idx = cell number
    int idy = blockIdx.y*blockDim.y + threadIdx.y;      // idy = local node number
    extern __shared__ double temp1[];

    if(idx < num_cells && idy < blockDim.y){
        // __device__ fn taken from other header to avoid code-reuse //
        assemble_elem(vertices, cells, is_bound, bdry_vals, temp1, idx, idy);
        __syncthreads();
        calc_weights(w, cells, temp1, idx, idy);
        elems_glob_cpy(Le, be, temp1, idx, idy);
    }
    if( (idx*3) + idy < order){
        u_glob[(idx*3) + idy] = 1.0;
    }
}
//////


/////////////// Kernel to calculate local approximation of solution ue /////////////
// Each cell has its own local solution for its element matrix and element vector
// These are apprimated with a jacobi iteration
__global__ void local_sols(
                double *Le,
                double *be,
                double *ue,
                double *up_glob,
                int *cells,
                int num_cells)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    extern __shared__ double temp1[]; 

    if(idx < num_cells && idy < blockDim.y){
        elems_shared_cpy(Le, be, temp1, idx, idy);
        __syncthreads();
        jacobi_iter(ue, up_glob, cells, temp1, idx, idy);
    }
}
///////


///////////// Kernel to calculate global approximation of u //////////////////
// Calculated by combining all local solutions ue with a weighting
__global__ void glob_sols(
                double *Le, 
                double *w, 
                double *u_glob, 
                double *ue, 
                int *cells,
                int num_cells)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int v;
    double Lii, weight;

    if(idx < num_cells && idy < blockDim.y){
        v = cells[(idx*3) + idy];               // getting global vertex number
        Lii = Le[(idx*9) + (idy*3) + idy];      
        
        weight = Lii/w[v];
        
        atomicAddDouble(&u_glob[v], weight * ue[(idx*3) + idy]);
    }
}
///////


/////////////// C++ function invoked to apply FEM-SES to solve PDE /////////////////////
// Applies the novel FEM - Single Element Solution approach to solve PDE
// Calculates element matrices as standard in regular approach
// Gets local solution approximations to these using a jacobi iteration
// Combines these to a global solution using a weighting
// Repeats until convergence of global solution
extern void gpu_femses(double *u, Mesh &M, Tau &t, int &count, int &reconfig){
    int nr[2];
    int order, num_cells;
    int block_size_Y, shared, shared2;
    double *vertices_gpu, *vertices;
    int *cells_gpu, *cells;
    int *dof_gpu, *dof;
    int *is_bound_gpu, *is_bound;
    double *bdry_vals_gpu, *bdry_vals;
    double *Le, *be, *ue, *w;
    double *up_gpu, *un_gpu;
    double err = 1E16;
    hipEvent_t start, finish, start2, finish2;
    hipError_t stat = hipSuccess;
    double tau = 0.0;
    int shrd_mem, threads;

    std::cout << GREEN "\nFEMSES Solver...\n" RESET;
    
    stat = hipSetDevice(k);
    assert(stat == hipSuccess);
    
    hipEventCreate(&start);
    hipEventCreate(&finish);
    
    //////////////////////////// Gathering info from mesh /////////////////////////////

    M.get_recs(nr);

    order = (nr[0]+1)*(nr[1]+1);
    num_cells = 2*nr[0]*nr[1];
    M.get_arrays(&vertices, &cells, &dof, &is_bound, &bdry_vals);

    ///////////////////////////////////////////////////////////////////////////////////
    
    
    ////////////// Allocating memory for mesh/stiffnesss matrix/stress vector//////////
    ///////////  /array of element matrics/array of stress vectors/weighting //////////

    hipEventRecord(start,0);

    stat = hipMalloc( (void**)&vertices_gpu, 2*order*sizeof(double));
    assert(stat == hipSuccess);
    stat = hipMalloc( (void**)&cells_gpu, 3*num_cells*sizeof(int));
    assert(stat == hipSuccess);
    stat = hipMalloc( (void**)&dof_gpu, 3*num_cells*sizeof(int));
    assert(stat == hipSuccess);
    stat = hipMalloc( (void**)&is_bound_gpu, order*sizeof(int));
    assert(stat == hipSuccess);
    stat = hipMalloc( (void**)&bdry_vals_gpu, order*sizeof(double));
    assert(stat == hipSuccess);

    stat = hipMalloc( (void**)&Le, num_cells*3*3*sizeof(double));
    assert(stat == hipSuccess);
    stat = hipMalloc( (void**)&be, num_cells*3*sizeof(double));
    assert(stat == hipSuccess);
    stat = hipMalloc( (void**)&ue, num_cells*3*sizeof(double));
    assert(stat == hipSuccess);
    stat = hipMalloc( (void**)&un_gpu, order*sizeof(double));
    assert(stat == hipSuccess);
    stat = hipMalloc( (void**)&up_gpu, order*sizeof(double));
    assert(stat == hipSuccess);
    stat = hipMalloc( (void**)&w, order*sizeof(double));
    assert(stat == hipSuccess);
    
    hipEventRecord(finish);
    hipEventSynchronize(finish);
    hipEventElapsedTime((float*)&t.alloc, start, finish);

    ///////////////////////////////////////////////////////////////////////////////////


    ///////////////// Copying data for Mesh from host to device ///////////////////////

    std::cout << "      Copying data from host...\n";
    
    hipEventRecord(start,0);

    stat = hipMemcpy(vertices_gpu, vertices, 2*order*sizeof(double), hipMemcpyHostToDevice);
    assert(stat == hipSuccess);
    stat = hipMemcpy(cells_gpu, cells, 3*num_cells*sizeof(int), hipMemcpyHostToDevice);
    assert(stat == hipSuccess);
    stat = hipMemcpy(dof_gpu, dof, 3*num_cells*sizeof(int), hipMemcpyHostToDevice);
    assert(stat == hipSuccess);
    stat = hipMemcpy(is_bound_gpu, is_bound, order*sizeof(int), hipMemcpyHostToDevice);
    assert(stat == hipSuccess);
    stat = hipMemcpy(bdry_vals_gpu, bdry_vals, order*sizeof(double), hipMemcpyHostToDevice);
    assert(stat == hipSuccess);
    
    hipEventRecord(finish);
    hipEventSynchronize(finish);
    hipEventElapsedTime((float*)&t.transfer, start, finish);

    stat = hipMemset(up_gpu, 0, order*sizeof(double));
    assert(stat == hipSuccess);

    ///////////////////////////////////////////////////////////////////////////////////


    //////////// DIMENSIONS OF SYSTEM => block per cell, 1 thread per node ////////////
    
    block_size_Y = 3;
    dim3 dimBlock(block_size_X, block_size_Y);
    dim3 dimGrid((num_cells/dimBlock.x)+(!(num_cells%dimBlock.x)?0:1),
            (1/dimBlock.y)+(!(1%dimBlock.y)?0:1));

    shared = 28*block_size_X;
    shared2 = 15*block_size_X;
   
    hipDeviceGetAttribute(&shrd_mem, hipDeviceAttributeMaxSharedMemoryPerBlock, k);
    hipDeviceGetAttribute(&threads, hipDeviceAttributeMaxThreadsPerBlock, k);

    // testing if shared memory is over the max amount on card //
    if(shared * sizeof(double) > shrd_mem){
        error_log();
        std::cerr << "      Not enough shared memory on device to continue..." << std::endl;
        std::cerr << "              Shared memory requested: " 
                                            << shared * sizeof(double) << std::endl;
        std::cerr << "              Shared memory available: " << shrd_mem << std::endl;
        std::cerr << "      Exiting." << std::endl;
        std::exit(1);
    }
    
    // testing if requested block size if over max amount allowed on card //
    if(block_size_X * block_size_Y > threads){
        std::exit(1);
        std::cerr << "      Too many threads requested per block..." << std::endl;
        std::cerr << "              Threads requested: " 
                                            << block_size_X * block_size_Y << std::endl;
        std::cerr << "              Max threads available: " << threads << std::endl;
        std::cerr << "      Exiting." << std::endl;
        error_log();
    }
    
    // reconfiguring memory if shared has spare, to allow more per thread registers //
    reconfig = 0;
    if(mem_config){ 
        if(shared * sizeof(double) < shrd_mem / 3){
            hipDeviceSetCacheConfig(hipFuncCachePreferL1);
            std::cout << "      Changed cache to prefer L1...\n";
            reconfig = 1;
        } else if(shared * sizeof(double) < shrd_mem / 2){
            hipDeviceSetCacheConfig(hipFuncCachePreferEqual);
            std::cout << "      Set cache to equal shared memory...\n";
            reconfig = 2;
        } else {
            hipDeviceSetCacheConfig(hipFuncCachePreferShared);
            reconfig = 0;
        }
    }

    //////////////////////////////////////////////////////////////////////////////////


    ////// Kernel to assemble element matrices and store in an array on glob mem /////

    std::cout << "      Getting element matrices...\n";
    
    hipEventRecord(start,0);
    
    assemble_elems_gpu<<<dimGrid, dimBlock, shared*sizeof(double)>>>
                (Le, be, w, up_gpu, vertices_gpu, cells_gpu, 
                 is_bound_gpu, bdry_vals_gpu, order, num_cells);

    hipEventRecord(finish);
    hipEventSynchronize(finish);
    hipEventElapsedTime((float*)&t.elem_mats, start, finish);

    //////////////////////////////////////////////////////////////////////////////////


    ///////////////// Iterates through kernels until convergence /////////////////////

    std::cout << "      Applying Jacobi relaxation scheme...\n";
    
    hipEventRecord(start,0);

    double *tmp;
    count = 0;
    while(err > EPS && count < MAX_ITERS){
        // getting local solutions ue and storing on global mem //
        local_sols<<<dimGrid, dimBlock, shared2*sizeof(double)>>>
                                    (Le, be, ue, up_gpu, cells_gpu, num_cells);
        
        // setting un_gpu to 0 //
        stat = hipMemset(un_gpu, 0, order*sizeof(double));
        assert(stat == hipSuccess);
        
        // assembling global solution estimate from weightings //
        glob_sols<<<dimGrid, dimBlock>>>(Le, w, un_gpu, ue, cells_gpu, num_cells);

        // calculating error using 2-norm //
        error_dot_prod(un_gpu, up_gpu, order, err);
        // std::cout << err << std::endl;

        tmp = un_gpu;
        un_gpu = up_gpu;
        up_gpu = tmp;

        count++;
        if(count == MAX_ITERS){
            std::cerr << "FEMSES - maximum iterations reached.\n";
            error_log();
            std::exit(1);
        }
    }

    hipEventRecord(finish);
    hipEventSynchronize(finish);
    hipEventElapsedTime((float*)&t.solve, start, finish);

    std::cout << "      Solved in " << count << " iterations...\n";
    
    //////////////////////////////////////////////////////////////////////////////////

    
    //////////////// Tranferring soln to host from device & tidy /////////////////////

    std::cout << "      Transferring result back to host...\n";
    
    hipEventRecord(start,0);
    
    hipMemcpy(u, up_gpu, order*sizeof(double), hipMemcpyDeviceToHost);
    
    hipEventRecord(finish);
    hipEventSynchronize(finish);
    hipEventElapsedTime((float*)&tau, start, finish);
    t.transfer += tau;

    hipFree(vertices_gpu);     hipFree(cells_gpu);    hipFree(dof_gpu);
    hipFree(is_bound_gpu);     hipFree(bdry_vals_gpu);
    hipFree(Le);               hipFree(be); 
    hipFree(un_gpu);           hipFree(up_gpu);
    hipFree(ue);               hipFree(w);

    //////////////////////////////////////////////////////////////////////////////////
}
////////
