#include "hip/hip_runtime.h"
#include <cassert>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipsolver.h>
#include <hipsparse.h>
#include "mesh.h"
#include "utils.h"
#include "gpu_utils.h"
#include "gpu_fem.h"

//////////// Calculates area of triangle, given coordinates ////////////////
__device__ float area(float *xi){
    float tmp = 0.0;

    tmp += xi[0]*(xi[4]*xi[8] - xi[5]*xi[7]);
    tmp -= xi[1]*(xi[3]*xi[8] - xi[5]*xi[6]);
    tmp += xi[2]*(xi[3]*xi[7] - xi[4]*xi[6]);

    return 0.5*tmp;
}
///////


///////////// Assembles element matrix and stress vector /////////////////////
// Calculated and stored on shared memory
__device__ void assemble_elem(
                float *vertices, 
                int *cells,  
                int *is_bound, 
                float *bdry_vals, 
                float *temp1, 
                int idx, 
                int idy)
{
    float *Le, *be, *xi, *consts;
    int v;
    float bound;

    Le = temp1;                 // element matrix
    be = &temp1[9];             // element stress vector
    xi = &temp1[12];            // matrix of global coordinates
    consts = &temp1[21];        // stores alpha, beta, gamma seen in serial version
    
    v = cells[(idx*3) + idy];   // global node number 
    
    //////////// Assigning global coordinates //////////////////
    
    xi[3*idy] = 1.0;
    xi[(3*idy) + 1] = vertices[2*v]; 
    xi[(3*idy) + 2] = vertices[(2*v)+1];

    __syncthreads();
    
    // using 1 thread to calculate area //
    if(idy==0)
        consts[9] = area(xi);
    
    ///////////////////////////////////////////////////////////


    //////////// calculating alpha, beta, gamma ///////////////
    
    // consts[(3*idy)] = xi[(idy+1)%3][1] * xi[(i+2)%3][2] - xi[(i+2)%3][1] * xi[(i+1)%3][2];
    consts[(3*idy)+1] = xi[ 3*((idy+1)%3) +2] - xi[ 3*((idy+2)%3) + 2];
    consts[(3*idy)+2] = xi[ 3*((idy+2)%3) +1] - xi[ 3*((idy+1)%3) + 1];
    
    __syncthreads();
    
    ////////////////////////////////////////////////////////////


    ///////////////////// Calculating LHS & RHS //////////////////

    be[idy] = 0.0;      // 0.0 intially until BCs enforced //
    for(int i=0; i<=idy; i++){
        Le[(3*idy)+i] = 0.25*consts[9]*consts[9]*consts[9] * (consts[(3*idy)+1]*consts[(3*i)+1] 
                                    + consts[(3*idy)+2]*consts[(3*i)+2]);
        Le[(3*i)+idy] = Le[(3*idy)+i];
    }
    __syncthreads();

    ///////////////////////////////////////////////////////////////


    ///////////////// Enforcing boundary conditions ///////////////
    
    if(is_bound[v]){
        bound = bdry_vals[v];
        // change this appropriately for more DOF if necessary //
        for(int j=0; j<3; j++){
            if(idy != j){
                atomicAdd(&be[j], (-1)*Le[(3*j) + idy]*bound);
                atomicExch(&Le[(3*j) + idy],  0.0);
                atomicExch(&Le[(3*idy) + j], 0.0);
            }
        }
        __syncthreads();
        Le[(3*idy)+idy] = 1.0;
        atomicExch(&be[idy], bound);
    }                            
    __syncthreads();
    
    /////////////////////////////////////////////////////////////////
}
////////


///////////////////// Assembles stiffness matrix and stress vector //////////
// Takes shared memory element matrices and vectors
// and maps them back to global memory as an assembled stiffness matrix
// Stored in dense format
__device__ void assemble_mat(
                float *L, 
                float *b, 
                float *vertices, 
                int *dof, 
                float *temp1, 
                int idx, 
                int idy, 
                int order)
{
    float *Le, *be;
    int* dof_r;
 
    Le = temp1;
    be = &temp1[9];
    dof_r = (int *)&temp1[12];      // stores in shared memory, global node numbers for 3 nodes

    ///////////////// Assigning global node numbers //////////////////

    if(idy==0){
        dof_r[0] = dof[(idx*3)];
        dof_r[1] = dof[(idx*3)+1];
        dof_r[2] = dof[(idx*3)+2];
    }
    __syncthreads();

    ////////////////////////////////////////////////////////////////
  

    ///////// Mapping values back to global stiffness matrix /////// 
    
    atomicAdd(&b[dof_r[idy]], be[idy]);
    
    for(int i=0; i<=idy; i++){
        atomicAdd(&L[(order*dof_r[idy]) + dof_r[i]], 
                            Le[(3*idy) + i]);
        if(i==idy)      // avoid double adding diagonal element
            continue;
        else {
            atomicAdd(&L[(order*dof_r[i]) + dof_r[idy]], 
                            Le[(3*idy) + i]);
        }
    }

    /////////////////////////////////////////////////////////////////
}
///////



///////////////////// Assembles stiffness matrix and stress vector //////////
// Takes shared memory element matrices and vectors
// and maps them back to global memory as an assembled stiffness matrix
// Stored in CSR format
__device__ void assemble_mat_csr(
                float *valsL, 
                int *rowPtrL, 
                int *colIndL, 
                float *b, 
                float *vertices, 
                int *dof, 
                float *temp1, 
                int idx, 
                int idy, 
                int order)
{
    float *Le, *be;
    int* dof_r;
    int row;
    int *tmp1, *tmp2;
    int off = 0;

    Le = temp1;
    be = &temp1[9];
    dof_r = (int *)&temp1[12];

    ///////////////// Assigning global node numbers //////////////////
    
    if(idy==0){
        dof_r[0] = dof[(idx*3)];
        dof_r[1] = dof[(idx*3)+1];
        dof_r[2] = dof[(idx*3)+2];
    }
    __syncthreads();
    
    //////////////////////////////////////////////////////////////////


    ///////// Mapping values back to global stiffness matrix ////////
    
    atomicAdd(&b[dof_r[idy]], be[idy]);
    
    row = rowPtrL[dof_r[idy]];
    tmp1 = &colIndL[row];
    for(int i=0; i<3; i++){
        tmp2 = tmp1;
        while(*tmp2 != dof_r[i]){
            off++;
            tmp2++;
        }
        atomicAdd(&valsL[row + off], Le[(3*idy) + i]);
        off = 0;
    }
    __syncthreads();

    ///////////////////////////////////////////////////////////////////
}
/////////


////////////// Kernel to calculate elements and assemble global stiffness matrix /////////////
// Assembled in dense format
__global__ void assemble_gpu(
                float *L, 
                float *b, 
                float *vertices, 
                int *cells, 
                int *is_bound, 
                float *bdry_vals, 
                int order)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;      // idx = cell number
    int idy = blockIdx.y*blockDim.y + threadIdx.y;      // idy = local node number
    extern __shared__ float temp1[];            // shared mem to store elem mats & constants

    if(idx < gridDim.x && idy < blockDim.y){
        assemble_elem(vertices, cells, is_bound, bdry_vals, temp1, idx, idy);
        assemble_mat(L, b, vertices, cells, temp1, idx, idy, order);
    }
}
///////



////////////// Kernel to calculate elements and assemble global stiffness matrix /////////////
// Assembled in CSR format
// see comments above
__global__ void assemble_gpu_csr(
                float *valsL, 
                int *rowPtrL, 
                int *colIndL, 
                float *b, 
                float *vertices, 
                int *cells, 
                int *is_bound, 
                float *bdry_vals, 
                int order)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    extern __shared__ float temp1[];

    if(idx < gridDim.x && idy < blockDim.y){
        assemble_elem(vertices, cells, is_bound, bdry_vals, temp1, idx, idy);
        assemble_mat_csr(valsL, rowPtrL, colIndL, b, vertices, cells, temp1, idx, idy, order);
    }
}    
////////


///////////////// C++ Function to be invoked from host to apply FEM to PDE ///////////////////
// Applies standard approach of assmebling stiffness matrix and
// decomposing the linear system on the GPU to solve
extern void gpu_fem(float *u, Mesh &M){
    int nr[2];
    int order, num_cells, nnz;
    int block_size_X, block_size_Y, shared;
    float *vertices_gpu, *vertices;
    int *cells_gpu, *cells;
    int *dof_gpu, *dof;
    int *is_bound_gpu, *is_bound;
    float *bdry_vals_gpu, *bdry_vals;
    float *L, *b, *valsL;
    int *rowPtrL, *colIndL;
    std::vector<float> valsLCPU;
    std::vector<int> rowPtrLCPU;
    std::vector<int> colIndLCPU;

    //////////////////////// Gathering info from mesh ////////////////////////////////

    M.get_recs(nr);

    order = (nr[0]+1)*(nr[1]+1);
    num_cells = 2*nr[0]*nr[1];
    M.get_arrays(&vertices, &cells, &dof, &is_bound, &bdry_vals);
    
    if(!dense) M.sparsity_pass(valsLCPU, rowPtrLCPU, colIndLCPU, nnz);

    //////////////////////////////////////////////////////////////////////////////////


    //////////// Allocating memory for mesh/stiffness matrix/stress vector //////////

    hipMalloc( (void**)&vertices_gpu, 2*order*sizeof(float));
    hipMalloc( (void**)&cells_gpu, 3*num_cells*sizeof(int));
    hipMalloc( (void**)&dof_gpu, 3*num_cells*sizeof(int));
    hipMalloc( (void**)&is_bound_gpu, order*sizeof(int));
    hipMalloc( (void**)&bdry_vals_gpu, order*sizeof(float));
    
    if(dense){
        hipMalloc( (void**)&L, order*order*sizeof(float));
    } else {
        hipMalloc( (void**)&valsL, nnz*sizeof(float)); 
        hipMalloc( (void**)&colIndL, nnz*sizeof(int)); 
        hipMalloc( (void**)&rowPtrL, (order+1)*sizeof(int)); 
    }
    hipMalloc( (void**)&b, order*sizeof(float));

    ////////////////////////////////////////////////////////////////////////////////


    /////////////// Copying data of Mesh from Host to Device ///////////////////////

    hipMemcpy(vertices_gpu, vertices, 2*order*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(cells_gpu, cells, 3*num_cells*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dof_gpu, dof, 3*num_cells*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(is_bound_gpu, is_bound, order*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(bdry_vals_gpu, bdry_vals, order*sizeof(float), hipMemcpyHostToDevice);
    
    ////////////////////////////////////////////////////////////////////////////////

   
    /////////////// Copying sparsity pattern if !dense ////////////////////////////
    
    if(!dense){
        hipMemcpy(rowPtrL, &rowPtrLCPU[0], (order+1)*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(colIndL, &colIndLCPU[0], nnz*sizeof(int), hipMemcpyHostToDevice);
    }

    ///////////////////////////////////////////////////////////////////////////////


    /////////  DIMENSIONS OF SYSTEM => 1 block per cell, 1 thread per node ////////
    
    block_size_X = 1, block_size_Y = 3;
    dim3 dimBlock(block_size_X, block_size_Y);
    dim3 dimGrid((num_cells/dimBlock.x)+(!(num_cells%dimBlock.x)?0:1),
                (1/dimBlock.y)+(!(1%dimBlock.y)?0:1));
    
    shared = 31;
    
    ///////////////////////////////////////////////////////////////////////////////

    
    //////// Kernel to assemble Stiffness matrix and store in global mem //////////

    if(dense) {
        assemble_gpu<<<dimGrid, dimBlock, shared*sizeof(float)>>>(L, b, vertices_gpu, 
                       cells_gpu, is_bound_gpu, bdry_vals_gpu, order);
    } else {
        assemble_gpu_csr<<<dimGrid, dimBlock, shared*sizeof(float)>>>(valsL, rowPtrL, colIndL, 
                        b, vertices_gpu, cells_gpu, is_bound_gpu, bdry_vals_gpu, order);
    }
    
    //////////////////////////////////////////////////////////////////////////////
     
    
    /////////// Solving linear system in dense, dense-sparse conversion, CSR format ////////

    if(dense){
        if(dnsspr)  dnsspr_solve(L, b, order);
        else        dense_solve(L, b, order);
    } else { 
        sparse_solve(valsL, rowPtrL, colIndL, b, order, nnz);
    }
    
    ///////////////////////////////////////////////////////////////////////////////////


    ////////////////// Transfer solution back to Host & tidy //////////////////////////

    hipMemcpy(u, b, order*sizeof(float), hipMemcpyDeviceToHost);

    
    hipFree(vertices_gpu);      hipFree(cells_gpu);   hipFree(dof_gpu);
    hipFree(is_bound_gpu);      hipFree(bdry_vals_gpu);
    if(dense)   hipFree(L),     hipFree(b);
    else        hipFree(valsL), hipFree(colIndL),     hipFree(rowPtrL); 

    /////////////////////////////////////////////////////////////////////////////////
}
