#include "hip/hip_runtime.h"
#include <cassert>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipsolver.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <vector>
#include "mesh.h"
#include "utils.h"
#include "gpu_utils.h"


////////////// Converts dense matrix to CSR and solves linear system ///////////////////
// function takes input of L, b and dim (order)
// Returns soln overwritten on b
// uses cuSolverSp and cuSparse
void dnsspr_solve(float *L, float *b, int order){
    hipsparseHandle_t handle = NULL;
    hipsolverSpHandle_t handleS = NULL;
    hipStream_t stream = NULL;
    hipsparseMatDescr_t desc = NULL;
    hipsparseStatus_t status = HIPSPARSE_STATUS_SUCCESS;
    hipsolverStatus_t status2 = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;                        
    const hipsparseDirection_t dir = HIPSPARSE_DIRECTION_ROW;     // row major

    int* csrRowPtrL = NULL;
    int* csrColIndL = NULL;
    float* csrValL  = NULL;
    int* nnzLrow;               // number of non zeros per row //
    int nnzL;
    const float err = EPS;
    int reorder = 0;
    int singularity;

    // Setting up streams //
    cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    assert(hipSuccess == cudaStat1);
    
    // creating cuSparse handle //
    status = hipsparseCreate(&handle);
    assert(HIPSPARSE_STATUS_SUCCESS == status);
    
    // creating cuSolver handle //
    status2 = hipsolverSpCreate(&handleS);
    assert(HIPSPARSE_STATUS_SUCCESS == status2);
    
    // setting stream to cuSparse //
    status = hipsparseSetStream(handle, stream);
    assert(HIPSPARSE_STATUS_SUCCESS == status);
    
    // creating matrixx description //
    status = hipsparseCreateMatDescr(&desc);
    assert(HIPSPARSE_STATUS_SUCCESS == status);
    
    /*  setting matrix description:
        0-base ordering, lower fill for Cholesky, general format */
    hipsparseSetMatIndexBase(desc, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(desc, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatFillMode(desc, HIPSPARSE_FILL_MODE_LOWER);
    
    // allocating memory for CSR arrays //
    cudaStat1 = hipMalloc( (void**)&csrRowPtrL, sizeof(int)*(order+1));
    assert(hipSuccess == cudaStat1);
    cudaStat1 = hipMalloc( (void**)&nnzLrow, sizeof(int)*(order));
    assert(hipSuccess == cudaStat1);
    
    // count number of no   n-zeros //
    status = hipsparseSnnz(handle, dir, order, order, desc, L, order, nnzLrow, &nnzL);
    assert(HIPSPARSE_STATUS_SUCCESS == status);
    
    // malloc remaining CSR array //
    cudaStat1 = hipMalloc( (void**)&csrValL, nnzL*sizeof(float));
    assert(hipSuccess == cudaStat1);
    cudaStat1 = hipMalloc( (void**)&csrColIndL, nnzL*sizeof(float));
    assert(hipSuccess == cudaStat1);

    // convert from dense to sparse //
    hipsparseSdense2csr(handle,order,order,desc,L,order,nnzLrow,csrValL,csrRowPtrL,csrColIndL);
    assert(HIPSPARSE_STATUS_SUCCESS == status);
    
    // set stream to cuSolver //
    status2 = hipsolverSpSetStream(handleS, stream);
    assert(HIPSOLVER_STATUS_SUCCESS == status2);

    // solver using Cholesky factorisation //
    status2 = hipsolverSpScsrlsvchol(handleS, order, nnzL, desc, csrValL, csrRowPtrL,
                                            csrColIndL, b, err, reorder, b, &singularity);
    assert(HIPSOLVER_STATUS_SUCCESS == status2);
    
    // destroy handles, desc & stream //
    hipsparseDestroy(handle);
    hipsolverSpDestroy(handleS);
    hipStreamDestroy(stream);
    hipsparseDestroyMatDescr(desc);
}
///////


///////////////////// Solves linear system in CSR format //////////////////////////
// NOTE: see comments from fn above //
void sparse_solve(float *valsL,int *rowPtrL, int *colPtrL, float *b, int order, int nnz){
    hipsolverSpHandle_t handle = NULL;
    hipStream_t stream = NULL;
    hipsparseMatDescr_t desc = NULL;
    hipsparseStatus_t status = HIPSPARSE_STATUS_SUCCESS;
    hipsolverStatus_t status2 = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;

    const float err = EPS;
    int reorder = 0;
    int singularity;

    cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    assert(hipSuccess == cudaStat1);
    
    status2 = hipsolverSpCreate(&handle);
    assert(HIPSOLVER_STATUS_SUCCESS == status2);

    status = hipsparseCreateMatDescr(&desc);
    assert(HIPSPARSE_STATUS_SUCCESS == status);
    
    hipsparseSetMatIndexBase(desc, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(desc, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatFillMode(desc, HIPSPARSE_FILL_MODE_LOWER);
    
    status2 = hipsolverSpSetStream(handle, stream);
    assert(HIPSOLVER_STATUS_SUCCESS == status2);
    
    status2 = hipsolverSpScsrlsvchol(handle, order, nnz, desc, valsL, rowPtrL,
                                            colPtrL, b, err, reorder, b, &singularity);
    //assert(HIPSOLVER_STATUS_EXECUTION_FAILED  == status2);
    assert(HIPSOLVER_STATUS_SUCCESS == status2);
    
    hipsolverSpDestroy(handle);
    hipStreamDestroy(stream);
    hipsparseDestroyMatDescr(desc);
}
////////


//////////////////// Solves linear system stored in dense format ////////////////////
// function takes input of L, b and dim (order)
// Returns soln overwritten on b
// uses cuSolverDn
void dense_solve(float *L, float *b, int order){
    hipsolverHandle_t handle = NULL;
    hipStream_t stream = NULL;
    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess; 
    const hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    const int nrhs = 1;
    float *buffer = NULL;
    int bufferSize = 0; 
    int *info = NULL;
    int h_info = 0;
    
    // setting cuSolver (Dense) handle //
    status = hipsolverDnCreate(&handle);
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    
    cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    assert(hipSuccess == cudaStat1);
    
    status = hipsolverSetStream(handle, stream);
    assert(HIPSOLVER_STATUS_SUCCESS == status); 

    // calculating buffer size needed for factorisation fn //
    status = hipsolverDnSpotrf_bufferSize(handle, uplo, order, L, order, &bufferSize);
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    
    // allocating space for buffer on GPU //
    cudaStat1 = hipMalloc( (void**)&info, sizeof(int));
    assert(hipSuccess == cudaStat1);
    cudaStat1 = hipMalloc( (void**)&buffer, bufferSize*sizeof(float));
    assert(hipSuccess == cudaStat1);
    hipMemset(info, 0, sizeof(int));

    // applying Cholesky factorisation to matrix //
    status = hipsolverDnSpotrf(handle, uplo, order, L, order, buffer, bufferSize, info);
    cudaStat1 = hipDeviceSynchronize();        // sync needed since non-blocking streams
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    assert(hipSuccess == cudaStat1);
    
    hipMemcpy(&h_info, info, sizeof(int), hipMemcpyDeviceToHost);

    // solving linear system - overwrites existing b //
    status = hipsolverDnSpotrs(handle, uplo, order, nrhs, L, order, b, order, info);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    assert(hipSuccess == cudaStat1);
      
    hipsolverDnDestroy(handle);
    hipStreamDestroy(stream);
}
///////


/////////////// Function gets error using 2-norm /////////////////
// Calculated using cuBLAS 
void error_dot_prod(float *a, float *b, int n, float &x){
    hipblasHandle_t handle;
    hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;
    const float alpha = -1.0;

    // creating cuBLAS handle //
    status = hipblasCreate(&handle);
    assert(status == HIPBLAS_STATUS_SUCCESS);
 
    // y = ax + y
    // function sets b = b - a
    status = hipblasSaxpy(handle, n, &alpha, a, 1, b, 1); 
    assert(status == HIPBLAS_STATUS_SUCCESS);

    // gets <b,b> //
    status = hipblasSnrm2(handle, n, b, 1, &x);
    assert(status == HIPBLAS_STATUS_SUCCESS);
    
    // destroys handle //
    status = hipblasDestroy(handle);
    assert(status == HIPBLAS_STATUS_SUCCESS);
}
///////


/////////////// Function gets max value of array /////////////////
// Calculated using cuBLAS
void array_max(float *a, int n, float &max){
    hipblasHandle_t handle;
    hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;
    
    // creating cuBLAS handle //
    status = hipblasCreate(&handle);
    assert(status == HIPBLAS_STATUS_SUCCESS);

    // getting maximum value of array //
    status = hipblasIsamax(handle, n, a, 0, &max);
    assert(status == HIPBLAS_STATUS_SUCCESS);
    
    // destroys handle //
    status = hipblasDestroy(handle);
    assert(status == HIPBLAS_STATUS_SUCCESS);
}
////////
