#include <cassert>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipsolver.h>
#include <hipsparse.h>
#include <hipblas.h>
#include "gpu_utils.h"

void dnsspr_solve(float *L, float *b, int order){
    hipsparseHandle_t handle = NULL;
    hipsolverSpHandle_t handleS = NULL;
    hipStream_t stream = NULL;
    hipsparseMatDescr_t desc = NULL;
    hipsparseStatus_t status = HIPSPARSE_STATUS_SUCCESS;
    hipsolverStatus_t status2 = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    const hipsparseDirection_t dir = HIPSPARSE_DIRECTION_ROW;

    int* csrRowPtrL = NULL;
    int* csrColIndL = NULL;
    float* csrValL  = NULL;
    int* nnzLrow;
    int nnzL;
    const float err = 1E-6;
    int reorder = 0;
    int singularity;

    cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    assert(hipSuccess == cudaStat1);
    
    status = hipsparseCreate(&handle);
    assert(HIPSPARSE_STATUS_SUCCESS == status);
    std::cout << "lefence issue here\n";
    
    status2 = hipsolverSpCreate(&handleS);
    assert(HIPSPARSE_STATUS_SUCCESS == status2);
    
    status = hipsparseSetStream(handle, stream);
    assert(HIPSPARSE_STATUS_SUCCESS == status);
    
    status = hipsparseCreateMatDescr(&desc);
    assert(HIPSPARSE_STATUS_SUCCESS == status);
    
    hipsparseSetMatIndexBase(desc, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(desc, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatFillMode(desc, HIPSPARSE_FILL_MODE_LOWER);
    
    cudaStat1 = hipMalloc( (void**)&csrRowPtrL, sizeof(int)*(order+1));
    assert(hipSuccess == cudaStat1);
    cudaStat1 = hipMalloc( (void**)&nnzLrow, sizeof(int)*(order));
    assert(hipSuccess == cudaStat1);
    
    status = hipsparseSnnz(handle, dir, order, order, desc, L, order, nnzLrow, &nnzL);
    assert(HIPSPARSE_STATUS_SUCCESS == status);
    
    cudaStat1 = hipMalloc( (void**)&csrValL, nnzL*sizeof(float));
    assert(hipSuccess == cudaStat1);
    cudaStat1 = hipMalloc( (void**)&csrColIndL, nnzL*sizeof(float));
    assert(hipSuccess == cudaStat1);

    hipsparseSdense2csr(handle,order,order,desc,L,order,nnzLrow,csrValL,csrRowPtrL,csrColIndL);

    status2 = hipsolverSpSetStream(handleS, stream);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    status2 = hipsolverSpScsrlsvchol(handleS, order, nnzL, desc, csrValL, csrRowPtrL,
                                            csrColIndL, b, err, reorder, b, &singularity);
    
    hipsparseDestroy(handle);
    hipsolverSpDestroy(handleS);
    hipStreamDestroy(stream);
    hipsparseDestroyMatDescr(desc);
}

void sparse_solve(float *valsL,int *rowPtrL, int *colPtrL, float *b, int order, int nnz){
    hipsolverSpHandle_t handle = NULL;
    hipStream_t stream = NULL;
    hipsparseMatDescr_t desc = NULL;
    hipsparseStatus_t status = HIPSPARSE_STATUS_SUCCESS;
    hipsolverStatus_t status2 = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;

    const float err = 1E-6;
    int reorder = 0;
    int singularity;

    cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    assert(hipSuccess == cudaStat1);
    
    status2 = hipsolverSpCreate(&handle);
    assert(HIPSPARSE_STATUS_SUCCESS == status2);
    
    status = hipsparseCreateMatDescr(&desc);
    assert(HIPSPARSE_STATUS_SUCCESS == status);
    
    hipsparseSetMatIndexBase(desc, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(desc, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatFillMode(desc, HIPSPARSE_FILL_MODE_LOWER);
    
    status2 = hipsolverSpSetStream(handle, stream);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    status2 = hipsolverSpScsrlsvchol(handle, order, nnz, desc, valsL, rowPtrL,
                                            colPtrL, b, err, reorder, b, &singularity);
    
    hipsolverSpDestroy(handle);
    hipStreamDestroy(stream);
    hipsparseDestroyMatDescr(desc);
}

void dense_solve(float *L, float *b, int order){
    hipsolverHandle_t handle = NULL;
    hipStream_t stream = NULL;
    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess; 
    const hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    const int nrhs = 1;
    float *Workspace;
    int Lwork, devInfo;

    status = hipsolverDnCreate(&handle);
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    std::cout << "lefence issue here\n";
    
    cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    assert(hipSuccess == cudaStat1);
    
    status = hipsolverSetStream(handle, stream);
    assert(HIPSOLVER_STATUS_SUCCESS == status); 

    status = hipsolverDnSpotrf_bufferSize(handle, uplo, order, L, order, &Lwork);
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    hipMalloc( (void**)&Workspace, Lwork*sizeof(float));

    status = hipsolverDnSpotrf(handle, uplo, order, L, order, Workspace, Lwork, &devInfo);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    assert(hipSuccess == cudaStat1);
    
    status = hipsolverDnSpotrs(handle, uplo, order, nrhs, L, order, b, order, &devInfo);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    assert(hipSuccess == cudaStat1);
      
    hipsolverDnDestroy(handle);
    hipStreamDestroy(stream);
}

void dotProd(float *a, float *b, int n, float &x){
    hipblasHandle_t handle;
    hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;
    
    status = hipblasCreate(&handle);
    assert(status == HIPBLAS_STATUS_SUCCESS);

    status = hipblasSdot(handle, n, a, 1, b, 1, &x); 
    assert(status == HIPBLAS_STATUS_SUCCESS);

    status = hipblasDestroy(handle);
    assert(status == HIPBLAS_STATUS_SUCCESS); 
}
