#include <cassert>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipsolver.h>
#include <hipsparse.h>
#include <hipblas.h>
#include "gpu_utils.h"

void dnsspr_solve(float *L, float *b, int order){
    hipsparseHandle_t handle = NULL;
    hipsolverSpHandle_t handleS = NULL;
    hipStream_t stream = NULL;
    hipsparseMatDescr_t desc = NULL;
    hipsparseStatus_t status = HIPSPARSE_STATUS_SUCCESS;
    hipsolverStatus_t status2 = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    const hipsparseDirection_t dir = HIPSPARSE_DIRECTION_ROW;

    int* csrRowPtrL = NULL;
    int* csrColIndL = NULL;
    float* csrValL  = NULL;
    int* nnzLrow;
    int nnzL;
    const float err = 1E-6;
    int reorder = 0;
    int singularity;

    cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    assert(hipSuccess == cudaStat1);
    
    status = hipsparseCreate(&handle);
    assert(HIPSPARSE_STATUS_SUCCESS == status);
    std::cout << "lefence issue here\n";
    
    status2 = hipsolverSpCreate(&handleS);
    assert(HIPSPARSE_STATUS_SUCCESS == status2);
    
    status = hipsparseSetStream(handle, stream);
    assert(HIPSPARSE_STATUS_SUCCESS == status);
    
    status = hipsparseCreateMatDescr(&desc);
    assert(HIPSPARSE_STATUS_SUCCESS == status);
    
    hipsparseSetMatIndexBase(desc, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(desc, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatFillMode(desc, HIPSPARSE_FILL_MODE_LOWER);
    
    cudaStat1 = hipMalloc( (void**)&csrRowPtrL, sizeof(int)*(order+1));
    assert(hipSuccess == cudaStat1);
    cudaStat1 = hipMalloc( (void**)&nnzLrow, sizeof(int)*(order));
    assert(hipSuccess == cudaStat1);
    
    status = hipsparseSnnz(handle, dir, order, order, desc, L, order, nnzLrow, &nnzL);
    assert(HIPSPARSE_STATUS_SUCCESS == status);
    
    cudaStat1 = hipMalloc( (void**)&csrValL, nnzL*sizeof(float));
    assert(hipSuccess == cudaStat1);
    cudaStat1 = hipMalloc( (void**)&csrColIndL, nnzL*sizeof(float));
    assert(hipSuccess == cudaStat1);

    hipsparseSdense2csr(handle,order,order,desc,L,order,nnzLrow,csrValL,csrRowPtrL,csrColIndL);

    status2 = hipsolverSpSetStream(handleS, stream);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    status2 = hipsolverSpScsrlsvchol(handleS, order, nnzL, desc, csrValL, csrRowPtrL,
                                            csrColIndL, b, err, reorder, b, &singularity);
    
    hipsparseDestroy(handle);
    hipsolverSpDestroy(handleS);
    hipStreamDestroy(stream);
    hipsparseDestroyMatDescr(desc);
}

void sparse_solve(float *valsL,int *rowPtrL, int *colPtrL, float *b, int order, int nnz){
    hipsolverSpHandle_t handle = NULL;
    hipStream_t stream = NULL;
    hipsparseMatDescr_t desc = NULL;
    hipsparseStatus_t status = HIPSPARSE_STATUS_SUCCESS;
    hipsolverStatus_t status2 = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;

    const float err = 1E-6;
    int reorder = 0;
    int singularity;

    cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    assert(hipSuccess == cudaStat1);
    
    status2 = hipsolverSpCreate(&handle);
    std::cout << status2 << std::endl;
    assert(HIPSPARSE_STATUS_SUCCESS == status2);

    status = hipsparseCreateMatDescr(&desc);
    assert(HIPSPARSE_STATUS_SUCCESS == status);
    
    hipsparseSetMatIndexBase(desc, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(desc, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatFillMode(desc, HIPSPARSE_FILL_MODE_LOWER);
    
    status2 = hipsolverSpSetStream(handle, stream);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    status2 = hipsolverSpScsrlsvchol(handle, order, nnz, desc, valsL, rowPtrL,
                                            colPtrL, b, err, reorder, b, &singularity);
    
    hipsolverSpDestroy(handle);
    hipStreamDestroy(stream);
    hipsparseDestroyMatDescr(desc);
}

void dense_solve(float *L, float *b, int order){
    hipsolverHandle_t handle = NULL;
    hipStream_t stream = NULL;
    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess; 
    const hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    const int nrhs = 1;
    float *buffer = NULL;
    int bufferSize = 0; 
    int *info = NULL;
    int h_info = 0;
    
    std::cout << "Dense testing not sparse\n";
    status = hipsolverDnCreate(&handle);
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    std::cout << "lefence issue here\n";
    
    cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    //cudaStat1 = hipStreamCreate(&stream);
    assert(hipSuccess == cudaStat1);
    
    status = hipsolverSetStream(handle, stream);
    assert(HIPSOLVER_STATUS_SUCCESS == status); 

    status = hipsolverDnSpotrf_bufferSize(handle, uplo, order, L, order, &bufferSize);
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    
    hipMalloc( (void**)&info, sizeof(int));
    hipMalloc( (void**)&buffer, bufferSize*sizeof(float));
    hipMemset(info, 0, sizeof(int));

    status = hipsolverDnSpotrf(handle, uplo, order, L, order, buffer, bufferSize, info);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    assert(hipSuccess == cudaStat1);
    
    hipMemcpy(&h_info, info, sizeof(int), hipMemcpyDeviceToHost);

    status = hipsolverDnSpotrs(handle, uplo, order, nrhs, L, order, b, order, info);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    assert(hipSuccess == cudaStat1);
      
    hipsolverDnDestroy(handle);
    hipStreamDestroy(stream);
}

void error_dot_prod(float *a, float *b, int n, float &x){
    hipblasHandle_t handle;
    hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;
    const float alpha = -1.0;
    float *res;

    hipMalloc( (void**)&res, n*sizeof(float));
    hipMemcpy(res, b, n*sizeof(float), hipMemcpyDeviceToDevice);

    status = hipblasCreate(&handle);
    assert(status == HIPBLAS_STATUS_SUCCESS);
 
    status = hipblasSaxpy(handle, n, &alpha, a, 1, res, 1); 
    assert(status == HIPBLAS_STATUS_SUCCESS);

    status = hipblasSnrm2(handle, n, res, 1, &x);
    assert(status == HIPBLAS_STATUS_SUCCESS);
    /*
    std::cout << "x " << x <<std::endl;
    status = hipblasSdot(handle, n, a, 1, b, 1, &x); 
    assert(status == HIPBLAS_STATUS_SUCCESS);
    std::cout << "x " << x <<std::endl;
    */
    status = hipblasDestroy(handle);
    assert(status == HIPBLAS_STATUS_SUCCESS); 
}
