#include "hip/hip_runtime.h"
#include <cstdio>
#include <cassert>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipsolver.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <vector>
#include "mesh.h"
#include "utils.h"
#include "gpu_utils.h"


////////////// Converts dense matrix to CSR and solves linear system ///////////////////
// function takes input of L, b and dim (order)
// Returns soln overwritten on b
// uses cuSolverSp and cuSparse
void dnsspr_solve(
            float *L,
            float *b,
            int order,
            hipEvent_t start,
            hipEvent_t finish,
            float &tau)
{
    hipsparseHandle_t handle = NULL;
    hipsolverSpHandle_t handleS = NULL;
    hipStream_t stream = NULL;
    hipsparseMatDescr_t desc = NULL;
    hipsparseStatus_t status = HIPSPARSE_STATUS_SUCCESS;
    hipsolverStatus_t status2 = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;                        
    const hipsparseDirection_t dir = HIPSPARSE_DIRECTION_ROW;     // row major

    int* csrRowPtrL = NULL;
    int* csrColIndL = NULL;
    float* csrValL  = NULL;
    int* nnzLrow;               // number of non zeros per row //
    int nnzL;
    const float err = EPS;
    int reorder = 0;
    int singularity;

    // Setting up streams //
    cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    assert(hipSuccess == cudaStat1);
    
    // creating cuSparse handle //
    status = hipsparseCreate(&handle);
    assert(HIPSPARSE_STATUS_SUCCESS == status);
    
    // creating cuSolver handle //
    status2 = hipsolverSpCreate(&handleS);
    assert(HIPSPARSE_STATUS_SUCCESS == status2);
    
    // setting stream to cuSparse //
    status = hipsparseSetStream(handle, stream);
    assert(HIPSPARSE_STATUS_SUCCESS == status);
    
    // creating matrixx description //
    status = hipsparseCreateMatDescr(&desc);
    assert(HIPSPARSE_STATUS_SUCCESS == status);
    
    /*  setting matrix description:
        0-base ordering, lower fill for Cholesky, general format */
    hipsparseSetMatIndexBase(desc, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(desc, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatFillMode(desc, HIPSPARSE_FILL_MODE_LOWER);
    
    hipEventRecord(start,0);    
    // allocating memory for CSR arrays //
    cudaStat1 = hipMalloc( (void**)&csrRowPtrL, sizeof(int)*(order+1));
    assert(hipSuccess == cudaStat1);
    cudaStat1 = hipMalloc( (void**)&nnzLrow, sizeof(int)*(order));
    assert(hipSuccess == cudaStat1);
    
    // count number of no   n-zeros //
    status = hipsparseSnnz(handle, dir, order, order, desc, L, order, nnzLrow, &nnzL);
    assert(HIPSPARSE_STATUS_SUCCESS == status);
    
    // malloc remaining CSR array //
    cudaStat1 = hipMalloc( (void**)&csrValL, nnzL*sizeof(float));
    assert(hipSuccess == cudaStat1);
    cudaStat1 = hipMalloc( (void**)&csrColIndL, nnzL*sizeof(float));
    assert(hipSuccess == cudaStat1);

    // convert from dense to sparse //
    hipsparseSdense2csr(handle,order,order,desc,L,order,nnzLrow,csrValL,csrRowPtrL,csrColIndL);
    assert(HIPSPARSE_STATUS_SUCCESS == status);
    hipEventRecord(finish, 0);
    hipEventSynchronize(finish);
    hipEventElapsedTime(&tau, start, finish);

    // set stream to cuSolver //
    status2 = hipsolverSpSetStream(handleS, stream);
    assert(HIPSOLVER_STATUS_SUCCESS == status2);

    // solver using Cholesky factorisation //
    status2 = hipsolverSpScsrlsvchol(handleS, order, nnzL, desc, csrValL, csrRowPtrL,
                                            csrColIndL, b, err, reorder, b, &singularity);
    assert(HIPSOLVER_STATUS_SUCCESS == status2);
    
    // destroy handles, desc & stream //
    hipsparseDestroy(handle);
    hipsolverSpDestroy(handleS);
    hipStreamDestroy(stream);
    hipsparseDestroyMatDescr(desc);
}
///////


///////////////////// Solves linear system in CSR format //////////////////////////
// NOTE: see comments from fn above //
void sparse_solve(
            float *valsL,
            int *rowPtrL,
            int *colPtrL,
            float *b,
            int order,
            int nnz)
{
    hipsolverSpHandle_t handle = NULL;
    hipStream_t stream = NULL;
    hipsparseMatDescr_t desc = NULL;
    hipsparseStatus_t status = HIPSPARSE_STATUS_SUCCESS;
    hipsolverStatus_t status2 = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;

    const float err = EPS;
    int reorder = 0;
    int singularity;

    cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    assert(hipSuccess == cudaStat1);
    
    status2 = hipsolverSpCreate(&handle);
    assert(HIPSOLVER_STATUS_SUCCESS == status2);

    status = hipsparseCreateMatDescr(&desc);
    assert(HIPSPARSE_STATUS_SUCCESS == status);
    
    hipsparseSetMatIndexBase(desc, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(desc, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatFillMode(desc, HIPSPARSE_FILL_MODE_LOWER);
    
    status2 = hipsolverSpSetStream(handle, stream);
    assert(HIPSOLVER_STATUS_SUCCESS == status2);
    
    status2 = hipsolverSpScsrlsvchol(handle, order, nnz, desc, valsL, rowPtrL,
                                            colPtrL, b, err, reorder, b, &singularity);
    //assert(HIPSOLVER_STATUS_EXECUTION_FAILED  == status2);
    assert(HIPSOLVER_STATUS_SUCCESS == status2);
    
    hipsolverSpDestroy(handle);
    hipStreamDestroy(stream);
    hipsparseDestroyMatDescr(desc);
}
////////


//////////////////// Solves linear system stored in dense format ////////////////////
// function takes input of L, b and dim (order)
// Returns soln overwritten on b
// uses cuSolverDn
void dense_solve(float *L, float *b, int order){
    hipsolverHandle_t handle = NULL;
    hipStream_t stream = NULL;
    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess; 
    const hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    const int nrhs = 1;
    float *buffer = NULL;
    int bufferSize = 0; 
    int *info = NULL;
    int h_info = 0;
    
    // setting cuSolver (Dense) handle //
    status = hipsolverDnCreate(&handle);
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    
    cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    assert(hipSuccess == cudaStat1);
    
    status = hipsolverSetStream(handle, stream);
    assert(HIPSOLVER_STATUS_SUCCESS == status); 

    // calculating buffer size needed for factorisation fn //
    status = hipsolverDnSpotrf_bufferSize(handle, uplo, order, L, order, &bufferSize);
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    
    // allocating space for buffer on GPU //
    cudaStat1 = hipMalloc( (void**)&info, sizeof(int));
    assert(hipSuccess == cudaStat1);
    cudaStat1 = hipMalloc( (void**)&buffer, bufferSize*sizeof(float));
    assert(hipSuccess == cudaStat1);
    hipMemset(info, 0, sizeof(int));

    // applying Cholesky factorisation to matrix //
    status = hipsolverDnSpotrf(handle, uplo, order, L, order, buffer, bufferSize, info);
    cudaStat1 = hipDeviceSynchronize();        // sync needed since non-blocking streams
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    assert(hipSuccess == cudaStat1);
    
    hipMemcpy(&h_info, info, sizeof(int), hipMemcpyDeviceToHost);

    // solving linear system - overwrites existing b //
    status = hipsolverDnSpotrs(handle, uplo, order, nrhs, L, order, b, order, info);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    assert(hipSuccess == cudaStat1);
      
    hipsolverDnDestroy(handle);
    hipStreamDestroy(stream);
}
///////


/////////////// Function gets error using 2-norm /////////////////
// Calculated using cuBLAS 
void error_dot_prod(float *a, float *b, int n, float &x){
    hipblasHandle_t handle;
    hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;
    const float alpha = -1.0;

    // creating cuBLAS handle //
    status = hipblasCreate(&handle);
    assert(status == HIPBLAS_STATUS_SUCCESS);
 
    // y = ax + y
    // function sets b = b - a
    status = hipblasSaxpy(handle, n, &alpha, a, 1, b, 1); 
    assert(status == HIPBLAS_STATUS_SUCCESS);

    // gets <b,b> //
    status = hipblasSnrm2(handle, n, b, 1, &x);
    assert(status == HIPBLAS_STATUS_SUCCESS);
    
    // destroys handle //
    status = hipblasDestroy(handle);
    assert(status == HIPBLAS_STATUS_SUCCESS);
}
///////


/////////////// Function gets max value of array /////////////////
// Calculated using cuBLAS
void array_max(double *a, int n, int &max){
    hipblasHandle_t handle;
    hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;
    
    // creating cuBLAS handle //
    status = hipblasCreate(&handle);
    assert(status == HIPBLAS_STATUS_SUCCESS);

    // getting maximum value of array //
    status = hipblasIdamax(handle, n, a, 0, &max);
    assert(status == HIPBLAS_STATUS_SUCCESS);
    
    // destroys handle //
    status = hipblasDestroy(handle);
    assert(status == HIPBLAS_STATUS_SUCCESS);
}
////////



////////////////////// Dummy kernel ///////////////////////////
// To run to reduce the effect of the initial
// kernel running slowly
__global__ void dummy_kernel(int n){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int count = 0;

    if(idx < n && idy < n){
        for(int i=0;i<n;i++)
            count++;
    }
}
//////


////////////////////// Dummy kernel ///////////////////////////
// To run to reduce the effect of the initial
// kernel running slowly
extern void dummy(float *dat, int n){
    float *a, *b, *c, *d, *e, *f;
    hipError_t stat = hipSuccess;

    stat = hipSetDevice(k);
    assert(stat == hipSuccess);
    
    stat = hipMalloc( (void**)&a, n*sizeof(float));
    assert(stat == hipSuccess);
    stat = hipMalloc( (void**)&b, n*sizeof(float));
    assert(stat == hipSuccess);
    stat = hipMalloc( (void**)&c, n*sizeof(float));
    assert(stat == hipSuccess);
    stat = hipMalloc( (void**)&d, n*sizeof(float));
    assert(stat == hipSuccess);
    stat = hipMalloc( (void**)&d, n*sizeof(float));
    assert(stat == hipSuccess);
    stat = hipMalloc( (void**)&e, n*sizeof(float));
    assert(stat == hipSuccess);
    stat = hipMalloc( (void**)&f, n*sizeof(float));
    assert(stat == hipSuccess);

    stat = hipMemcpy(a, dat, n*sizeof(float), hipMemcpyHostToDevice);
    assert(stat == hipSuccess);
    stat = hipMemcpy(b, dat, n*sizeof(float), hipMemcpyHostToDevice);
    assert(stat == hipSuccess);
    stat = hipMemcpy(c, dat, n*sizeof(float), hipMemcpyHostToDevice);
    assert(stat == hipSuccess);
    stat = hipMemcpy(d, dat, n*sizeof(float), hipMemcpyHostToDevice);
    assert(stat == hipSuccess);
    stat = hipMemcpy(e, dat, n*sizeof(float), hipMemcpyHostToDevice);
    assert(stat == hipSuccess);
    stat = hipMemcpy(f, dat, n*sizeof(float), hipMemcpyHostToDevice);
    assert(stat == hipSuccess);

    dim3 dimBlock(50, 10);
    dim3 dimGrid((n/dimBlock.x) + (!(n%dimBlock.x)?0:1),
                (n/dimBlock.y) + (!(n%dimBlock.y)?0:1));

    dummy_kernel<<<dimGrid, dimBlock>>>(n);

    hipFree(a);    hipFree(b);    hipFree(c);
    hipFree(d);    hipFree(e);    hipFree(f);
}
///////
