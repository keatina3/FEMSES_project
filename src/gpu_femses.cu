#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <cassert>
#include <cstdio>
#include <vector>
#include "mesh.h"
#include "utils.h"
#include "gpu_utils.h"
#include "gpu_fem.h"
#include "gpu_femses.h"

//////////// Calculates weighting for assembling single element solution ///////////
// One weight is evaluated for each node
// Added back to global memory
__device__ void calc_weights(float *w, int *cells, float *temp1, int idx, int idy){
    float *Le;
    int v;
    int offset = 28*threadIdx.x;

    Le = &temp1[offset];
    v = cells[(idx*3) + idy];

    atomicAdd(&w[v], Le[(idy*3) + idy]);
}
////////


/////////// Copies element matrices/element vector from global-shared memory //////////
__device__ void elems_glob_cpy(float *Le, float *be, float *temp1, int idx, int idy){
    float *Le_shrd, *be_shrd;
    int offset = 28*threadIdx.x;

    Le_shrd = &temp1[offset];
    be_shrd = &temp1[offset + 9];

    be[(idx*3) + idy] = be_shrd[idy];

    for(int i=0; i<3; i++){
        Le[(idx*9) + (idy*3) + i] = Le_shrd[(idy*3) + i];
    }
}
////////


/////////// Copies element matrices/element vector from shared-global memory //////////
__device__ void elems_shared_cpy(float *Le, float *be, float *temp1, int idx, int idy){
    float *Le_shrd, *be_shrd;
    int offset = 15*threadIdx.x;

    Le_shrd = &temp1[offset];
    be_shrd = &temp1[offset + 9];

    be_shrd[idy] = be[(idx*3) + idy];
    for(int i=0; i<3; i++){
        Le_shrd[(idy*3) + i] = Le[(idx*9) + (idy*3) + i];
    }
}
////////


////////////// Performs Jacobi iteration to get updated approximation of u ////////////
__device__ void jacobi_iter(
                float *ue,
                float *up_glob,
                int *cells,
                float *temp1,
                int idx,
                int idy)
{
    float *Le_shrd, *be_shrd;
    float ue_new, *ue_old;
    int v;
    int offset = 15*threadIdx.x;

    Le_shrd = &temp1[offset];
    be_shrd = &temp1[offset + 9];
    ue_old  = &temp1[offset + 12];

    v = cells[(idx*3) + idy];

    ue_new = be_shrd[idy];
    ue_old[idy] = up_glob[v];

    __syncthreads();
    
    ue_new -= Le_shrd[(idy*3) + ((idy+1)%3) ] * ue_old[ (idy+1) % 3];
    ue_new -= Le_shrd[(idy*3) + ((idy+2)%3) ] * ue_old[ (idy+2) % 3];

    ue_new /= Le_shrd[(idy*3) + idy];

    atomicExch(&ue[(idx*3) + idy], ue_new); // transferring element solution of u to global mem
}
//////


/////////////////// Kernel to assemble element solutions ///////////////////////////
// Element solutions are calculated in shared memory
// Element solutions are then transferred to an array in global memory
__global__ void assemble_elems_gpu(
                float *Le, 
                float *be, 
                float *w,
                float *u_glob,
                float *vertices, 
                int *cells, 
                int *is_bound, 
                float *bdry_vals,
                int order,
                int num_cells)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;      // idx = cell number
    int idy = blockIdx.y*blockDim.y + threadIdx.y;      // idy = local node number
    extern __shared__ float temp1[];

    if(idx < num_cells && idy < blockDim.y){
        // __device__ fn taken from other header to avoid code-reuse //
        assemble_elem(vertices, cells, is_bound, bdry_vals, temp1, idx, idy);
        __syncthreads();
        calc_weights(w, cells, temp1, idx, idy);
        elems_glob_cpy(Le, be, temp1, idx, idy);
    }
    if( (idx*3) + idy < order){
        u_glob[(idx*3) + idy] = 1.0;
    }
}
//////


/////////////// Kernel to calculate local approximation of solution ue /////////////
// Each cell has its own local solution for its element matrix and element vector
// These are apprimated with a jacobi iteration
__global__ void local_sols(
                float *Le,
                float *be,
                float *ue,
                float *up_glob,
                int *cells,
                int num_cells)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    extern __shared__ float temp1[]; 

    if(idx < num_cells && idy < blockDim.y){
        elems_shared_cpy(Le, be, temp1, idx, idy);
        __syncthreads();
        jacobi_iter(ue, up_glob, cells, temp1, idx, idy);
    }
}
///////


///////////// Kernel to calculate global approximation of u //////////////////
// Calculated by combining all local solutions ue with a weighting
__global__ void glob_sols(
                float *Le, 
                float *w, 
                float *u_glob, 
                float *ue, 
                int *cells,
                int num_cells)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    int v;
    float Lii, weight;

    if(idx < num_cells && idy < blockDim.y){
        v = cells[(idx*3) + idy];               // getting global vertex number
        Lii = Le[(idx*9) + (idy*3) + idy];      
        
        weight = Lii/w[v];
        
        atomicAdd(&u_glob[v], weight * ue[(idx*3) + idy]);
    }
}
///////


/////////////// C++ function invoked to apply FEM-SES to solve PDE /////////////////////
// Applies the novel FEM - Single Element Solution approach to solve PDE
// Calculates element matrices as standard in regular approach
// Gets local solution approximations to these using a jacobi iteration
// Combines these to a global solution using a weighting
// Repeats until convergence of global solution
extern void gpu_femses(float *u, Mesh &M, Tau &t){
    int nr[2];
    int order, num_cells;
    int block_size_Y, shared, shared2;
    float *vertices_gpu, *vertices;
    int *cells_gpu, *cells;
    int *dof_gpu, *dof;
    int *is_bound_gpu, *is_bound;
    float *bdry_vals_gpu, *bdry_vals;
    float *Le, *be, *ue, *w;
    float *up_gpu, *un_gpu;
    float err = 1E16;
    hipEvent_t start, finish, start2, finish2;
    hipError_t stat = hipSuccess;
    float tau = 0.0;
    int shrd_mem, threads;

    std::cout << GREEN "\nFEMSES Solver...\n" RESET;
    
    hipSetDevice(k);
    
    hipEventCreate(&start);
    hipEventCreate(&finish);
    
    //////////////////////////// Gathering info from mesh /////////////////////////////

    M.get_recs(nr);

    order = (nr[0]+1)*(nr[1]+1);
    num_cells = 2*nr[0]*nr[1];
    M.get_arrays(&vertices, &cells, &dof, &is_bound, &bdry_vals);

    ///////////////////////////////////////////////////////////////////////////////////
    
    
    ////////////// Allocating memory for mesh/stiffnesss matrix/stress vector//////////
    ///////////  /array of element matrics/array of stress vectors/weighting //////////

    hipEventRecord(start,0);

    stat = hipMalloc( (void**)&vertices_gpu, 2*order*sizeof(float));
    assert(stat == hipSuccess);
    stat = hipMalloc( (void**)&cells_gpu, 3*num_cells*sizeof(int));
    assert(stat == hipSuccess);
    stat = hipMalloc( (void**)&dof_gpu, 3*num_cells*sizeof(int));
    assert(stat == hipSuccess);
    stat = hipMalloc( (void**)&is_bound_gpu, order*sizeof(int));
    assert(stat == hipSuccess);
    stat = hipMalloc( (void**)&bdry_vals_gpu, order*sizeof(float));
    assert(stat == hipSuccess);

    stat = hipMalloc( (void**)&Le, num_cells*3*3*sizeof(float));
    assert(stat == hipSuccess);
    stat = hipMalloc( (void**)&be, num_cells*3*sizeof(float));
    assert(stat == hipSuccess);
    stat = hipMalloc( (void**)&ue, num_cells*3*sizeof(float));
    assert(stat == hipSuccess);
    stat = hipMalloc( (void**)&un_gpu, order*sizeof(float));
    assert(stat == hipSuccess);
    stat = hipMalloc( (void**)&up_gpu, order*sizeof(float));
    assert(stat == hipSuccess);
    stat = hipMalloc( (void**)&w, order*sizeof(float));
    assert(stat == hipSuccess);
    
    hipEventRecord(finish);
    hipEventSynchronize(finish);
    hipEventElapsedTime(&t.alloc, start, finish);

    ///////////////////////////////////////////////////////////////////////////////////


    ///////////////// Copying data for Mesh from host to device ///////////////////////

    std::cout << "      Copying data from host...\n";
    
    hipEventRecord(start,0);

    stat = hipMemcpy(vertices_gpu, vertices, 2*order*sizeof(float), hipMemcpyHostToDevice);
    assert(stat == hipSuccess);
    stat = hipMemcpy(cells_gpu, cells, 3*num_cells*sizeof(int), hipMemcpyHostToDevice);
    assert(stat == hipSuccess);
    stat = hipMemcpy(dof_gpu, dof, 3*num_cells*sizeof(int), hipMemcpyHostToDevice);
    assert(stat == hipSuccess);
    stat = hipMemcpy(is_bound_gpu, is_bound, order*sizeof(int), hipMemcpyHostToDevice);
    assert(stat == hipSuccess);
    stat = hipMemcpy(bdry_vals_gpu, bdry_vals, order*sizeof(float), hipMemcpyHostToDevice);
    assert(stat == hipSuccess);
    
    hipEventRecord(finish);
    hipEventSynchronize(finish);
    hipEventElapsedTime(&t.transfer, start, finish);

    stat = hipMemset(up_gpu, 0, order*sizeof(float));
    assert(stat == hipSuccess);

    ///////////////////////////////////////////////////////////////////////////////////


    //////////// DIMENSIONS OF SYSTEM => block per cell, 1 thread per node ////////////
    
    block_size_Y = 3;
    dim3 dimBlock(block_size_X, block_size_Y);
    dim3 dimGrid((num_cells/dimBlock.x)+(!(num_cells%dimBlock.x)?0:1),
            (1/dimBlock.y)+(!(1%dimBlock.y)?0:1));

    shared = 28*block_size_X;
    shared2 = 15*block_size_X;
   
    hipDeviceGetAttribute(&shrd_mem, hipDeviceAttributeMaxSharedMemoryPerBlock, k);
    hipDeviceGetAttribute(&threads, hipDeviceAttributeMaxThreadsPerBlock, k);

    if(shared * sizeof(float) > shrd_mem){
        std::cerr << "      Not enough shared memory on device to continue..." << std::endl;
        std::cerr << "              Shared memory requested: " 
                                            << shared * sizeof(float) << std::endl;
        std::cerr << "              Shared memory available: " << shrd_mem << std::endl;
        std::cerr << "      Exiting." << std::endl;
        std::exit(1);
    }

    if(block_size_X * block_size_Y > threads){
        std::cerr << "      Too many threads requested per block..." << std::endl;
        std::cerr << "              Threads requested: " 
                                            << block_size_X * block_size_Y << std::endl;
        std::cerr << "              Max threads available: " << threads << std::endl;
        std::cerr << "      Exiting." << std::endl;
        std::exit(1);
    }

    //////////////////////////////////////////////////////////////////////////////////


    ////// Kernel to assemble element matrices and store in an array on glob mem /////

    std::cout << "      Getting element matrices...\n";
    
    hipEventRecord(start,0);
    
    assemble_elems_gpu<<<dimGrid, dimBlock, shared*sizeof(float)>>>
                (Le, be, w, up_gpu, vertices_gpu, cells_gpu, 
                 is_bound_gpu, bdry_vals_gpu, order, num_cells);

    hipEventRecord(finish);
    hipEventSynchronize(finish);
    hipEventElapsedTime(&t.elem_mats, start, finish);

    //////////////////////////////////////////////////////////////////////////////////


    ///////////////// Iterates through kernels until convergence /////////////////////

    std::cout << "      Applying Jacobi relaxation scheme...\n";
    
    hipEventRecord(start,0);

    float *tmp;
    int count = 0;
    while(err > EPS && count < MAX_ITERS){
        // getting local solutions ue and storing on global mem //
        local_sols<<<dimGrid, dimBlock, shared2*sizeof(float)>>>
                                    (Le, be, ue, up_gpu, cells_gpu, num_cells);
        
        // setting un_gpu to 0 //
        stat = hipMemset(un_gpu, 0, order*sizeof(float));
        assert(stat == hipSuccess);
        
        // assembling global solution estimate from weightings //
        glob_sols<<<dimGrid, dimBlock>>>(Le, w, un_gpu, ue, cells_gpu, num_cells);

        // calculating error using 2-norm //
        error_dot_prod(un_gpu, up_gpu, order, err);
        // std::cout << err << std::endl;

        tmp = un_gpu;
        un_gpu = up_gpu;
        up_gpu = tmp;

        count++;
        if(count == MAX_ITERS){
            std::cerr << "FEMSES - maximum iterations reached.\n";
            exit(1);
        }
    }

    hipEventRecord(finish);
    hipEventSynchronize(finish);
    hipEventElapsedTime(&t.solve, start, finish);

    std::cout << "      Solved in " << count << " iterations...\n";
    
    //////////////////////////////////////////////////////////////////////////////////

    
    //////////////// Tranferring soln to host from device & tidy /////////////////////

    std::cout << "      Transferring result back to host...\n";
    
    hipEventRecord(start,0);
    
    hipMemcpy(u, up_gpu, order*sizeof(float), hipMemcpyDeviceToHost);
    
    hipEventRecord(finish);
    hipEventSynchronize(finish);
    hipEventElapsedTime(&tau, start, finish);
    t.transfer += tau;

    hipFree(vertices_gpu);     hipFree(cells_gpu);    hipFree(dof_gpu);
    hipFree(is_bound_gpu);     hipFree(bdry_vals_gpu);
    hipFree(Le);               hipFree(be); 
    hipFree(un_gpu);           hipFree(up_gpu);
    hipFree(ue);               hipFree(w);

    //////////////////////////////////////////////////////////////////////////////////
}
////////
